#include "hip/hip_runtime.h"
#include "../utils/DataReader.h"
#include "../data_split/DataMerge.h"
#include <vtkm/cont/Initialize.h>
#include <iostream>
#include "../compress/CompressorMGARD_nonCLI.h"
#include <chrono>
#include <cmath> // for std::sqrt
#include <algorithm> // for std::max

int main(int argc, char *argv[]) {
    vtkm::cont::InitializeOptions options =
            vtkm::cont::InitializeOptions::RequireDevice | vtkm::cont::InitializeOptions::AddHelp;
    vtkm::cont::Initialize(argc, argv, options);

    // std::string filePath = "../data/100x500x500/Pf48.bin.f32";
    std::string filePath = "../data/SDRBENCH-EXASKY-NYX-512x512x512/temperature.f32";
    size_t numElements = 512 * 512 * 512;
    std::vector<vtkm::Float32> data = readF32File<vtkm::Float32>(filePath, numElements);

    vtkm::Id3 dataDimensions(500, 500, 100);
    vtkm::Id3 blockDimensions(16, 16, 16);
    int numIsovalues = 5;

    try {
        auto mergedBlocks = findAndMergeIsosurfaceBlocks<vtkm::Float32>(data, dataDimensions, blockDimensions, numIsovalues);

        std::cout << "Merged block positions, sizes, and dimensions: " << std::endl;
        for (const auto &block: mergedBlocks) {
            const auto &mergedData = std::get<0>(block);
            const auto &position = std::get<1>(block);
            const auto &dimensions = std::get<2>(block);
        }

        
        double errorBound = 4e-4;
        std::string s = "infinity"; // Setting s to infinity for L-infinity norm
        double totalCompressionTime = 0;
        double totalDecompressionTime = 0;
        double totalCompressedSize = 0;
        double totalOriginalSize = 0;

        // double totalL2AbsError = 0;
        // double totalL2RelError = 0;
        double totalLinfAbsError = 0;
        double totalLinfRelError = 0;

        for (const auto &block: mergedBlocks) {
            const auto &mergedData = std::get<0>(block);
            const auto &dimensions = std::get<2>(block);

            auto start = std::chrono::high_resolution_clock::now();
            CompressionResult mgardCompressed = compressDataWithMGARDX(mergedData, dimensions[0], dimensions[1], dimensions[2], errorBound, s);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> compressionTime = end - start;
            totalCompressionTime += compressionTime.count();
            totalCompressedSize += mgardCompressed.compressedData.size();
            totalOriginalSize += mergedData.size() * sizeof(float);

            start = std::chrono::high_resolution_clock::now();
            std::vector<float> decompressedVec = decompressDataWithMGARDX(mgardCompressed.compressedData, dimensions[0], dimensions[1], dimensions[2]);
            end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> decompressionTime = end - start;
            totalDecompressionTime += decompressionTime.count();

            // float l2_abs_error = calculate_L2_error(mergedData, decompressedVec, false);
            // float l2_rel_error = calculate_L2_error(mergedData, decompressedVec, true);
            float linf_abs_error = calculate_Linf_error(mergedData, decompressedVec, false);
            float linf_rel_error = calculate_Linf_error(mergedData, decompressedVec, true);

            // totalL2AbsError += l2_abs_error * l2_abs_error * mergedData.size();
            // totalL2RelError += l2_rel_error * l2_rel_error * mergedData.size();
            totalLinfAbsError = std::max(totalLinfAbsError, static_cast<double>(linf_abs_error));
            totalLinfRelError = std::max(totalLinfRelError, static_cast<double>(linf_rel_error));

            // std::cout << "L2 Absolute Error: " << l2_abs_error << std::endl;
            // std::cout << "L2 Relative Error: " << l2_rel_error << std::endl;
            // std::cout << "L-infinity Absolute Error: " << linf_abs_error << std::endl;
            // std::cout << "L-infinity Relative Error: " << linf_rel_error << std::endl;
        }

        // totalL2AbsError = std::sqrt(totalL2AbsError / totalOriginalSize);
        // totalL2RelError = std::sqrt(totalL2RelError / totalOriginalSize);

        std::cout << "Total Compression Time: " << totalCompressionTime << " seconds" << std::endl;
        std::cout << "Total Decompression Time: " << totalDecompressionTime << " seconds" << std::endl;
        std::cout << "Total Compressed Size: " << totalCompressedSize << " bytes" << std::endl;
        std::cout << "Total Original Size: " << totalOriginalSize << " bytes" << std::endl;
        std::cout << "Compression Ratio: " << totalOriginalSize / totalCompressedSize << std::endl;

        // std::cout << "Total L2 Absolute Error: " << totalL2AbsError << std::endl;
        // std::cout << "Total L2 Relative Error: " << totalL2RelError << std::endl;
        std::cout << "Total L-infinity Absolute Error: " << totalLinfAbsError << std::endl;
        std::cout << "Total L-infinity Relative Error: " << totalLinfRelError << std::endl;

    } catch (const std::exception &e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }

    return 0;
}

